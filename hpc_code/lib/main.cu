#include "hip/hip_runtime.h"
//
//  main.cpp
//  diffusion_membrane_exchange
//
//  Update Journal:
//  -- 11/14/2019: re-write cylinder code to membrane code
//  -- 11/17/2019: membrane thickness = 0
//
//  Created by Hong-Hsi Lee in November, 2019.
//


#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <math.h>
#include <iomanip>
#include <time.h>
#include <cstdlib>
#include <algorithm>
#include <string>
#include <complex>
#include <string>
//#include "diffusion_lib.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;
    
#define Pi 3.14159265
#define timepoints 10000

// ********** diffusion kernel **********
__device__ void pixPosition ( const double &x_in, const unsigned int &NPix, int &xPix ) {
    double x=x_in;
    if ( x<0 ) { x+=1; }
    if ( x>1 ) { x-=1; }
    xPix=floor(x*NPix);
}

__device__ void translateXc ( const double &x, double &xc ) {
    // Translate circle center xc to make it as close to the position x as possible
    int ii=2*(xc<0.5)-1;
    double ti=0, d2=fabs(x-xc), d2Tmp=fabs(x-xc-ii);
    if (d2Tmp<d2) { ti=ii; }
    xc+=ti;
}

__device__ void crossmembrane( const double &xi, const double &xf, const double &xc_in, const bool &translateFlag, bool &instruction) {
    double xc=xc_in;
    if (translateFlag) { translateXc(xf,xc); }
    instruction = (xc-xi)*(xc-xf)<=0;
}

__device__ void elastic(const double &x, const double &v, const double &dx, const double &xc_in, const bool &translateFlag, double &xt) {
    double xc=xc_in;
    if (translateFlag) { translateXc(x,xc); }
    if (fabs(x-xc)<dx) {
        xt=xc-v*(dx-fabs(x-xc));
    }
}

// ********** cuda kernel **********
__device__ double atomAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
        
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    
    return __longlong_as_double(old);
}

__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state, unsigned long seed){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void propagate(hiprandStatePhilox4_32_10_t *state, double *dx2, double *dx4, double *sig, const int TN, const int NPar, const double res, const double dt, const double stepIN, const double probI, const unsigned int NPix, const int initFlag, const double *xCir, const bool *translateFlag, const int Nbval, const double *bval, const unsigned int *APix){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandStatePhilox4_32_10_t localstate=state[idx];
    
    int Tstep=TN/timepoints;
    
    for (int k=idx; k<NPar; k+=stride){
        double xPar=0, xCirTmp=0;
        
        unsigned int a=0, aTmp=0;                           // Element of APix matrix
        unsigned int acell[2]={0}; bool instruction[2]={0}; // Cell label
        
        double xi=0, xt=0, xTmp=0;                          // Particle position
        int xtG=0, xTmpG=0;                                 // Position on grid after diffusion
        double vrand=0;                                     // Random number
        int tidx=0;
        
        double vp=0;                                        // Nomalized diffusion velocity
        int acell_hit=0;                                    // Label of the cell encountered by the walker
        
        double xjmp=0;
        double dx=0, TD=0, qx=0;
        
        //********** Initialize Walker Positions *********
        while (1){
            xPar=hiprand_uniform_double(&localstate);
            
            if ( initFlag==1 ) { // 1. Initial positon: ICS
                xi=xPar;
                break;
            } else if ( initFlag==2 ) { // 2. Initial position: center
                xi=0.5;
                break;
            }
        }
        
        // ********** Simulate diffusion **********
        xt=xi;
        for (int i=0; i<TN; i++){
            // The cells close to the walker in the previous step
            pixPosition(xt,NPix,xtG);                             // Position on grid
            a=APix[ xtG ];
            
            // ********** One step **********
            // Case 1 Diffusion In ICS
            acell[0]=a;
            // Primitive position after diffusion
            vrand=hiprand_uniform_double(&localstate);
            vp=2*(vrand>0.5)-1;
            xTmp=xt+stepIN*vp;
            
            pixPosition(xTmp,NPix,xTmpG);
            aTmp=APix[ xTmpG ];
            acell[1]=aTmp;
            
            // Check if the segment(xt,xTmp) overlaps with any membrane
            for (int j=0; j<2; j++) {
                if (acell[j]==0) {
                    instruction[j]=false;
                } else {
                    crossmembrane(xt,xTmp,xCir[acell[j]-1],translateFlag[acell[j]-1],instruction[j]);
                }
            }
            if ( (instruction[0]==false) & (instruction[1]==false) ) {
                // Case 1.1 Walker diffuses in ICS and does not encounter any cell membrane.
                xt=xTmp;
            } else {
                // Case 1.2 Walker diffuses in ICS and encounters the cell membrane.
                
                // Determine the cell to collide with.
                acell_hit=0;
                for (int j=0; j<2; j++) {
                    if ( instruction[j] ) { acell_hit=acell[j]; }
                }
                
                if ( acell_hit==0 ){
                    printf("error: Walker in ICS does not encounter the cell membrane.\n");
                }
                
                xCirTmp=xCir[acell_hit-1];
                
                vrand=hiprand_uniform_double(&localstate);
                if (vrand<probI) {
                    // Case 1.2.1 Permeation through membrane
                    xt=xTmp;
                }
                else {
                    // Case 1.2.2 Elastic collision in ECS
                    elastic(xt, vp, stepIN, xCirTmp, translateFlag[acell_hit-1], xTmp);
                    xt=xTmp;
                }
            }
            
            // Periodic boundary condition
            if (xt>1) {
                xt-=1;
                xjmp+=1;
            }
            else if (xt<0) {
                xt+=1;
                xjmp-=1;
            }
            
            // ********** End one step **********
            
            if ( (i%Tstep) ==0 ) { // Save moment tensor for dx^2 and dx^4, and signal for the b-table
                tidx=i/Tstep;
                
                dx=(xt+xjmp-xi)*res;
                atomAdd(&dx2[tidx],dx*dx);
                atomAdd(&dx4[tidx],dx*dx*dx*dx);
                
                TD=i*dt;
                for (int j=0; j<Nbval; j++) {
                    qx = sqrt(bval[j]/TD) * dx;
                    atomAdd(&sig[tidx*Nbval+j],cos(qx));
                }
            }
            
        }
    }
    state[idx]=localstate;
}

    
//********** Define tissue parameters **********

int main(int argc, char *argv[]) {
    
    clock_t begin=clock();
    clock_t end=clock();
    
    // Define index number
    int i=0, j=0;
    
    //********** Load mictostructure **********
    
    double dt=0;                // Time step in ms
    int TN=0;                   // Number of time steps
    int NPar=0;                 // Number of particles
    int Nbval=0;                // Number of b-values
    
    double Din=0;               // Diffusion coefficient inside the axon in �m^2/ms
    double kappa=0;             // Permeability of a lipid bi-layer in �m/ms
    int initFlag=1;             // Initial position: 1=ICS, 2=center
    int thread_per_block=0;     // Number of threads per block
    
    unsigned int NPix=0, NAx=0;
    double res=0;
    
    // simulation parameter
    ifstream myfile0 ("simParamInput.txt", ios::in);
    myfile0>>dt; myfile0>>TN; myfile0>>NPar;
    myfile0>>Din; myfile0>>kappa;
    myfile0>>initFlag;
    myfile0>>thread_per_block;
    myfile0.close();
    
    double stepIN=sqrt(2.0*dt*Din);     // Step size in ICS in �m
    
    // resolution
    ifstream myfile1 ("phantom_res.txt", ios::in);
    myfile1>>res;
    myfile1.close();
    
    // Pixel # along each side
    ifstream myfile2 ("phantom_NPix.txt", ios::in);
    myfile2>>NPix;
    myfile2.close();
    
    // Pixelized matrix A indicating axon labels
    thrust::host_vector<unsigned int> APix(NPix);
    ifstream myfile3 ("phantom_APix.txt", ios::in);
    for (i=0; i<NPix; i++){
        myfile3>>APix[i];
    }
    myfile3.close();
    
    // Number of axons
    ifstream myfile4 ("phantom_NMem.txt", ios::in);
    myfile4>>NAx;
    myfile4.close();
    
    // Circle center of x-coordinate
    thrust::host_vector<double> xCir(NAx);
    ifstream myfile5 ("phantom_xMem.txt", ios::in);
    for (i=0; i<NAx; i++){
        myfile5>>xCir[i];
    }
    myfile5.close();
    
    // Number of b-values
    ifstream myfile6 ("Nbval.txt", ios::in);
    myfile6>>Nbval;
    myfile6.close();
    
    // b-value
    thrust::host_vector<double> bval(Nbval);
    ifstream myfile7 ("bval.txt", ios::in);
    for (i=0; i<Nbval; i++){
        myfile7>>bval[i];
    }
    myfile7.close();
    
    //********** Initialize Particle Positions in IAS *********
    const double probI=stepIN*kappa/Din;          // Probability constant from ICS to myelin
    stepIN/=res;
    cout<<"probI="<<probI<<endl;
    cout<<"NPix="<<NPix<<endl;
    cout<<"NAx="<<NAx<<endl;
    // Create translate flag to speed up the code
    thrust::host_vector<bool> translateFlag(NAx);
    for (i=0; i<NAx; i++) {
        if ( (xCir[i]+2*stepIN>=1) | (xCir[i]-2*stepIN<=0) ) {
            translateFlag[i]=true;
        } else {
            translateFlag[i]=false;
        }
    }
    
    // ********** Simulate diffusion **********
    
    // Initialize seed
    unsigned long seed=0;
    FILE *urandom;
    urandom = fopen("/dev/random", "r");
    fread(&seed, sizeof (seed), 1, urandom);
    fclose(urandom);
    
    // Initialize state of RNG
    int blockSize = thread_per_block;
    int numBlocks = (NPar + blockSize - 1) / blockSize;
    cout<<numBlocks<<endl<<blockSize<<endl;
    
    thrust::device_vector<hiprandStatePhilox4_32_10_t> devState(numBlocks*blockSize);
    setup_kernel<<<numBlocks, blockSize>>>(devState.data().get(),seed);
    
    // Initialize output
    thrust::host_vector<double> dx2(timepoints);
    thrust::host_vector<double> dx4(timepoints);
    thrust::host_vector<double> sig(timepoints*Nbval);
    for (i=0;i<timepoints;i++){ dx2[i]=0; }
    for (i=0;i<timepoints;i++){ dx4[i]=0; }
    for (i=0;i<timepoints*Nbval;i++){ sig[i]=0; }
    
    // Move data from host to device
    thrust::device_vector<double> d_dx2=dx2;
    thrust::device_vector<double> d_dx4=dx4;
    thrust::device_vector<double> d_sig=sig;
    thrust::device_vector<double> d_xCir=xCir;
    thrust::device_vector<bool> d_translateFlag=translateFlag;
    thrust::device_vector<double> d_bval=bval;
    thrust::device_vector<unsigned int> d_APix=APix;
    
    // Parallel computation
    begin=clock();
    propagate<<<numBlocks, blockSize>>>(devState.data().get(), d_dx2.data().get(), d_dx4.data().get(), d_sig.data().get(), TN, NPar, res, dt, stepIN, probI, NPix, initFlag, d_xCir.data().get(), d_translateFlag.data().get(), Nbval, d_bval.data().get(), d_APix.data().get());
    hipDeviceSynchronize();
    end=clock();
    cout << "Done! Elpased time "<<double((end-begin)/CLOCKS_PER_SEC) << " s"<< endl;
    
    thrust::copy(d_dx2.begin(), d_dx2.end(), dx2.begin());
    thrust::copy(d_dx4.begin(), d_dx4.end(), dx4.begin());
    thrust::copy(d_sig.begin(), d_sig.end(), sig.begin());
    
    ofstream fdx2out("dx2_diffusion.txt");
    ofstream fdx4out("dx4_diffusion.txt");
    ofstream fsigout("sig_diffusion.txt");
    fdx2out.precision(15);
    fdx4out.precision(15);
    fsigout.precision(15);
    
    for (i=0; i<timepoints; i++) {
        fdx2out<<dx2[i]<<endl;
        fdx4out<<dx4[i]<<endl;
        for (j=0; j<Nbval; j++) {
            if ( j==(Nbval-1) ) {
                fsigout<<sig[i*Nbval+j]<<endl;
            } else {
                fsigout<<sig[i*Nbval+j]<<"\t";
            }
        }
    }
    fdx2out.close();
    fdx4out.close();
    fsigout.close();
    
    ofstream paraout ("sim_para.txt");
    paraout<<dt<<endl<<TN<<endl<<NPar<<endl;
    paraout<<Din<<endl;
    paraout<<kappa<<endl<<initFlag<<endl;
    paraout.close();
    
    ofstream TDout("diff_time.txt");
    for (i=0; i<timepoints; i++){
        TDout<<(i*(TN/timepoints)+1)*dt<<endl;
    }
    TDout.close();
}

